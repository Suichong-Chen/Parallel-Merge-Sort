#include "hip/hip_runtime.h"
#include "timerc.h"
#include <math.h>

__device__ __host__ void merge(int *a, int left_start, int right_end) { // a is source array
    int left1, right1, left2, right2, k, i, j;
    int size = 1;
    int len_arr = right_end - left_start + 1;
    int* b =  (int *) malloc(sizeof(int) * len_arr);    // b is temp array

    while (size < len_arr) {
        k = left_start;
        left1 = left_start;
        while (left1 + size < len_arr) {
            right1 = left1 + size - 1;
            left2 = right1 + 1;
            right2 = left2 + size - 1;
            if (right2 >= len_arr) {
                right2 = len_arr - 1;
            }

            // start merging the list
            i = left1;
            j = left2;
            while (i <= right1 && j <= right2) {
                if (a[i] < a[j]) {
                    b[k] = a[i];
                    i++;
                    k++;
                    //printf("a[i] = %d\n", a[i]);
                }
                else {
                    b[k] = a[j];
                    j++;
                    k++;
                    //printf("a[i] = %d\n", a[i]);
                }
            }

            // if either left or right still has remaining
            while (i <= right1) {
                b[k] = a[i];
                i++;
                k++;
            }
            while (j <= right2) {
                b[k] = a[j];
                j++;
                k++;
            }

            //merge and sort other pairs
            left1 = right2 + 1;
        }
        // if there is any pair left that is unmerged
        i = left1;
        while (k < len_arr) {
            b[k] = a[i];
            i++;
            k++;
        }

        for (i = 0; i < len_arr; i++) {
            a[i] = b[i];
        }

        size = size * 2;
    }
    free(b);
}

__global__ void gpu_mergesort_serial_merge(int *a, int level) {

    // if
    int thread_i  = threadIdx.x + blockIdx.x * blockDim.x;

    int left_start = pow(2,level) * 2 * thread_i;
    //int left_end = pow(2,level) * 2 * thread_i + pow(2,level) - 1;
    //int right_start = pow(2,level) * ( 2 * thread_i + 1 );
    int right_end = pow(2,level) * ( 2 * thread_i + 1 ) + pow(2,level) - 1;

    printf("this is left start = %d \n", a[left_start]);
    printf("this is right end = %d \n", a[right_end]);
    merge(a, left_start, right_end);
}


void test_merge(){

    int a[8] ={4,2,4,7,9,1,3,8};
    merge(a, 0, 7);

    for (int i = 0; i < 7; i++){
        printf("%d ", a[i]);
    }
    printf("\n");
}


void test_pow_2(){
    printf("%lf\n", pow(2,3));
}

void test_log_2(){
    printf("%lf\n", log2(2));
}

int main() {
    /* test functions
    test_log_2();
    test_pow_2();
    test_merge();
    */

    int n = 256;
    int num_threads_per_block = 64;
    int* h_arr =  (int *) malloc(sizeof(int)*n);

    //generate an array with numbers
    for (int i = 0; i < n; i++) {
        h_arr[i] = (n-1) - i;
    }

    //int size_arr = n;

    // call gpu_mergesort and generate gpu_result
    int * d_arr;
    //int * d_temp;
    int * gpu_result = (int *) malloc( n * sizeof(int) );
    hipMalloc( (void**) &d_arr, n * sizeof(int) );
    hipMemcpy( d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice );
    //hipMalloc( (void**) &d_temp, n * sizeof(int) );

    for (int lev = 0; lev < log2(n); lev++) {   // lev means level
        gpu_mergesort_serial_merge<<< n / num_threads_per_block / pow(2, lev+1) , num_threads_per_block >>>(d_arr, lev);
        // copy d_temp into d_arr
        //hipMemcpy(d_arr, d_temp, size_arr * sizeof(int), hipMemcpyDeviceToDevice);
    }

    /*
    hipMemcpy(d_temp, d_arr, size_arr * sizeof(int), hipMemcpyDeviceToDevice);
    int flag = 0;
    for (int lev = 0; lev <= log2(size); lev++) {   // lev means level
        if (flag == 0) {
            gpu_mergesort_serial_merge<<< num_threads_per_block , size_arr / (pow(2, (lev + 1)) * num_threads_per_block) >>>(d_arr, d_temp, lev);
            flag = 1;
        }
        else {
            gpu_mergesort_serial_merge<<< num_threads_per_block , size_arr / (pow(2, (lev + 1)) * num_threads_per_block) >>>(d_temp, d_arr, lev);
            flag = 0;
        }
        hipDeviceSynchronize();
    }

    if (flag == 1) {    // finish on d_temp
        hipMemcpy(gpu_result, d_temp, ( size_arr * sizeof(int) ), hipMemcpyDeviceToHost);
    }
    else {  // finish on d_arr
        hipMemcpy(gpu_result, d_arr, ( size_arr * sizeof(int) ), hipMemcpyDeviceToHost);
    }
*/
    hipMemcpy(gpu_result, d_arr, ( n * sizeof(int) ), hipMemcpyDeviceToHost);

    // ----------------------------------------------------------------------------------------

    /*
    // merge sort on CPU
    merge(h_arr, 0, n-1);
*/
    // debug
    printf("gpu result: \n");
    for (int i = 0; i < n; i++) {
        printf("%d ", gpu_result[i]);
    }
    printf("\n");
/*
    // debug
    printf("cpu result: \n");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    // compare cpu_result with gpu_result
    for (int i = 0; i < n; i++) {
        if (gpu_result[i] != h_arr[i]) {
            printf("ERROR\n");
            break;
        }
    }
    printf("gpu operation has the same result as the cpu operation\n");
*/
}